#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include "Core.h"
//#include "Generator.h"

using namespace cv;
using namespace std;

namespace fractal
{
    namespace core
    {
        __global__ static void kernel(unsigned char* buffer, int width, int height, int fractalType, double seedR, double seedI, double focusR, double focusI, double zoomWidth, unsigned int iterations, double escapeRadius, bool smooth,
            unsigned char* gradientColors, double* gradientKeyPositions, int gradientKeyCount, unsigned char fractalColorR, unsigned char fractalColorG, unsigned char fractalColorB)
        {
            long int pixelCount = width * height;
            int stride = blockDim.x * gridDim.x;

            double escapeRadiusSquared = escapeRadius * escapeRadius;

            double scalar = zoomWidth / width;

            double ofsetR = focusR - (width - 1) / 2 * scalar;
            double ofsetI = focusI - (height - 1) / 2 * scalar;

            for (long int pixel = (long int)blockIdx.x * blockDim.x + threadIdx.x; pixel < pixelCount; pixel += stride)
            {
                //screenspace coordinates
                int x = pixel % width;
                int y = height - 1 - (pixel - x) / width;

                //transform from screenspace to complex numbers
                double r = x * scalar + ofsetR;
                double i = y * scalar + ofsetI;

                //setup variables
                double Zr, Zi, Cr, Ci;
                if (fractalType == 0) {//!!!!!!!!!!!!!!! mandelbrotSet
                    Zr = seedR;
                    Zi = seedI;
                    Cr = r;
                    Ci = i;
                }
                else {
                    Zr = r;
                    Zi = i;
                    Cr = seedR;
                    Ci = seedI;
                }

                //iterations
                bool escapes = false;
                int I;
                for (I = 0; I < iterations; I++)
                {
                    //check if escaped
                    if (Zr * Zr + Zi * Zi >= escapeRadiusSquared) {
                        escapes = true;
                        break;
                    }

                    //square Z
                    double oldZr = Zr;
                    Zr = Zr * Zr - Zi * Zi;
                    Zi = 2 * oldZr * Zi;

                    //add C to Z
                    Zr += Cr;
                    Zi += Ci;
                }

                //calculate color
                unsigned char R;
                unsigned char G;
                unsigned char B;

                if (escapes)
                {
                    double key;
                    //if (smooth) {
                    //    //key = fmod(I - log2(log2(Zr * Zr + Zi * Zi)) + 4.0, gradientKeyPositions[gradientKeyCount - 1]);//modulus (gradient's length as base) of smoothed I
                    //    //key = I % (int)gradientKeyPositions[gradientKeyCount - 1];
                    //}
                    //else {
                    //    //key = I % (int)gradientKeyPositions[gradientKeyCount - 1];
                    //    key = fmod((double)I, (double)gradientKeyPositions[gradientKeyCount - 1]);//modulus of I
                    //}
                    //key = fmod((double)I, (double)gradientKeyPositions[gradientKeyCount - 1]);//modulus of I

                    key = I % (int)gradientKeyPositions[gradientKeyCount - 1];


                    int colorA, colorB;
                    double fraction;
                    for (int i = 1; i < gradientKeyCount; i++)
                    {
                        if (gradientKeyPositions[i] >= key) {
                            colorA = i - 1;
                            colorB = i;
                            fraction = (key - gradientKeyPositions[i - 1]) / (gradientKeyPositions[i] - gradientKeyPositions[i - 1]);
                            break;
                        }
                    }

                    R = gradientColors[colorA * 3 + 0] + (gradientColors[colorB * 3 + 0] - gradientColors[colorA * 3 + 0]) * fraction;
                    G = gradientColors[colorA * 3 + 1] + (gradientColors[colorB * 3 + 1] - gradientColors[colorA * 3 + 1]) * fraction;
                    B = gradientColors[colorA * 3 + 2] + (gradientColors[colorB * 3 + 2] - gradientColors[colorA * 3 + 2]) * fraction;
                }
                else {
                    R = fractalColorR;
                    G = fractalColorG;
                    B = fractalColorB;
                }

                //write to buffer
                buffer[3 * pixel] = B;
                buffer[3 * pixel + 1] = G;
                buffer[3 * pixel + 2] = R;
            }
        }

        bool GPUFrameBuffer::updateGradient(const unsigned char* const gradient, int gradientSize, double gradientStretch)//encode gradient if necessary (source values changed). returns true, if update was necessary
        {
            gradientSize = max(0, gradientSize);

            //check if update is necessary
            if (gradientStretch != this->gradientStretchComparer || gradientSize != this->gradientSizeComparer)
                goto Update;
            for (int i = 0; i < gradientSize; i++)
            {
                if (gradient[i] != this->gradientComparer[i])
                    goto Update;
            }
            return 0;//if all was same, don't update

        Update:
            //free old memory, except if this is first update
            if (gradientSizeComparer != -1) {
                hipFree(gradientColors);
                hipFree(gradientKeyPositions);
            }

            //update comparers
            gradientComparer = (unsigned char*)gradient;
            memcpy(gradientComparer, gradient, sizeof(unsigned char) * gradientSize);
            gradientSizeComparer = gradientSize;
            gradientStretchComparer = gradientStretch;

            //encode gradient (onto host). update gradientKeyCount
            gradientKeyCount = gradientSize / 4 + 1;//+1: also last color (same as first) included
            unsigned char* _gradientColors = new unsigned char[gradientKeyCount * 3];
            double* _gradientKeyPositions = new double[gradientKeyCount];
            _gradientKeyPositions[0] = 0;//first key is at 0
            for (int i = 0; i < gradientKeyCount - 1; i++)
            {
                _gradientColors[i * 3 + 0] = gradient[i * 4 + 0];
                _gradientColors[i * 3 + 1] = gradient[i * 4 + 1];
                _gradientColors[i * 3 + 2] = gradient[i * 4 + 2];

                _gradientKeyPositions[i + 1] = _gradientKeyPositions[i] + gradient[i * 4 + 3] * gradientStretch;
            }
            _gradientColors[gradientKeyCount - 3] = gradient[0];//last color is the same as first
            _gradientColors[gradientKeyCount - 2] = gradient[1];
            _gradientColors[gradientKeyCount - 1] = gradient[2];

            //copy to GPU
            hipMalloc((void**)&gradientColors, sizeof(unsigned char) * 3 * gradientKeyCount);
            hipMemcpy((void*)gradientColors, (void*)_gradientColors, sizeof(unsigned char) * 3 * gradientKeyCount, hipMemcpyHostToDevice);
            hipMalloc((void**)&gradientKeyPositions, sizeof(double) * gradientKeyCount);
            hipMemcpy((void*)gradientKeyPositions, (void*)_gradientKeyPositions, sizeof(double) * gradientKeyCount, hipMemcpyHostToDevice);

            //free host data
            delete[] _gradientColors;
            delete[] _gradientKeyPositions;

            return true;
        }

        GPUFrameBuffer::GPUFrameBuffer(int _width, int _height)
        {
            width = _width;
            height = _height;

            hipMallocManaged(&buffer, sizeof(unsigned char) * (long int)width * height * 3);
            encodedBuffer = Mat(height, width, CV_8UC3, buffer);
        }

        const Mat* GPUFrameBuffer::frame(int fractalType, double seedR, double seedI, double focusR, double focusI, double zoomWidth, unsigned int iterations, double escapeRadius, bool smooth,
            const unsigned char* const gradient, int gradientSize, double gradientStretch, const unsigned char* fractalColor)
        {
            updateGradient(gradient, gradientSize, gradientStretch);

            kernel<<<512, 512>>>(buffer, width, height, fractalType, seedR, seedI, focusR, focusI, zoomWidth, iterations, escapeRadius, smooth,
                this->gradientColors, this->gradientKeyPositions, this->gradientKeyCount, fractalColor[0], fractalColor[1], fractalColor[2]);
            hipDeviceSynchronize();

            return &encodedBuffer;
        }

        GPUFrameBuffer::~GPUFrameBuffer() {
            hipFree(buffer);
            encodedBuffer.release();

            if (gradientSizeComparer != -1) {
                hipFree(gradientColors);
                hipFree(gradientKeyPositions);
            }
        }
    }
}